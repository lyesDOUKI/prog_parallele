
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* vec, int threadsParBlock) {
	long index = blockIdx.x*threadsParBlock+threadIdx.x;
	vec[index] = vec[index] + 5;
}

int main(void) {
	int vectorSize = 256;
	int numberOfBlock = 32;
	int threadsParBlock = 8;

	int *vec_cpu = (int*)malloc(sizeof(int) * vectorSize);
	int *vec_gpu;

    hipMalloc((int**)&vec_gpu, sizeof(int) * vectorSize);

	for(int i = 0; i < vectorSize; i++) vec_cpu[i] = 10;

    hipMemcpy(vec_gpu, vec_cpu, sizeof(int) * vectorSize, hipMemcpyHostToDevice);

	add<<<numberOfBlock,threadsParBlock>>>(vec_gpu, threadsParBlock);

    hipMemcpy(vec_cpu, vec_gpu, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);

	for(int i = 0; i < vectorSize; i++) {
		printf("%d ", vec_cpu[i]);
	}
	printf("\n");


    return 0;
}