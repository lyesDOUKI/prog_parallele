#include "hip/hip_runtime.h"
 #include <stdio.h>
  2 
  3 __global__ void add(int* vec,int * vec2, int * vec3,  int threadsParBlock) {
  4     long index = blockIdx.x*threadsParBlock+threadIdx.x;
  5     vec3[index] = vec[index] + vec2[index];
  6 }
  7 
  8 int main(void) {
  9     int vectorSize = 5;
 10     int numberOfBlock = 32;
 11     int threadsParBlock = 8;
 12  
 13     int *vec_cpu = (int*)malloc(sizeof(int) * vectorSize);
 14     int *vec_gpu;
 15 
 16     int *vec_cpu2 = (int*)malloc(sizeof(int) * vectorSize);
 17     int *vec_gpu2;
 18 
 19     int *vec_cpu3 = (int*)malloc(sizeof(int) * vectorSize);
 20     int *vec_gpu3;
 21 
 22     hipMalloc((int**)&vec_gpu, sizeof(int) * vectorSize);
 23     hipMalloc((int**)&vec_gpu2, sizeof(int) * vectorSize);
 24     hipMalloc((int**)&vec_gpu3, sizeof(int) * vectorSize);
 25 
 26     for(int i = 0; i < vectorSize; i++) vec_cpu[i] = 3;
 27     for(int i = 0; i < vectorSize; i++) vec_cpu2[i] = 3;
 28     for(int i = 0; i < vectorSize; i++) vec_cpu3[i] = 0;
 29 
 30     hipMemcpy(vec_gpu, vec_cpu, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
 31     hipMemcpy(vec_gpu2, vec_cpu2, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
 32     hipMemcpy(vec_gpu3, vec_cpu3, sizeof(int) * vectorSize, hipMemcpyHostToDevice);
 33 
 34 
 35     add<<<numberOfBlock,threadsParBlock>>>(vec_gpu, vec_gpu2, vec_gpu3, threadsParBlock);
 36 
 37     hipMemcpy(vec_cpu3, vec_gpu3, sizeof(int) * vectorSize, hipMemcpyDeviceToHost);
 38 
 39     for(int i = 0; i < vectorSize; i++) {
 40         printf("%d ", vec_cpu3[i]);
 41     }
 42     printf("\n");
 43 
 44 
 45     return 0;
 46 }
~                                                                                                                                                                                                                                                            
~            